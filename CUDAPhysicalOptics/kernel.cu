#include "hip/hip_runtime.h"
//CUDA Core for Physical Optics Calculation
//v0.0, 2018.05
//Ming Jin jinmingaps@163.com
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#include "./hip/hip_runtime_api.h"
#include ""
#include "hip/device_functions.h"
#include <stdio.h>
#include "stdlib.h"
#include "stdio.h"
#include "stdlib.h"
#include "CUDAfunction.h"
#include "cuVector3.h"
#include "cuComplexVector3.h"
#include "Constant_Val.h"

//Total amount of constant memory:          65536 bytes			�����ڴ� �൱��ÿ���߳�128��Float��4byte�� 128threads
//Total amount of shared memory per block : 49152 bytes			�����ڴ� �൱��ÿ���߳�96��Float (4byte) 128threds
//Total number of registers available per block : 65536			�Ĵ��� �൱��ÿ���߳�128��Float (4byte) 128threds

//ÿ��Block��128���߳�
const int threadsPerBlock = 128;

#if CUDART_VERSION < 5000

// CUDA-C includes
#include <hip/hip_runtime.h>


// This function wraps the CUDA Driver API into a template function
#include
template <class T>
inline void getCudaAttribute(T *attribute, hipDeviceAttribute_t device_attribute, int device)
{
	hipError_t error = hipDeviceGetAttribute(attribute, device_attribute, device);

	if (hipSuccess != error)
	{
		fprintf(stderr, "cuSafeCallNoSync() Driver API error = %04d from file <%s>, line %i.\n",
			error, __FILE__, __LINE__);

		exit(EXIT_FAILURE);
	}
}

#endif /* CUDART_VERSION < 5000 */

__global__ void //�����������������0 _ȷ����������������ҲΪ0_�����ļ�������һ���ж�
kernel_SetZero(cuVector3* _d_p_in, float* _d_ds, cuComplexVector3* _d_J_in) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	//int shift = blockDim.x * blockIdx.x;
	int ii = threadIdx.x;
	_d_p_in[i] = SetcuVector3(100, 100, 100);
	_d_ds[i] = 0;
	_d_J_in[i] = SetcuComplexVector3f(0.0, 0.0, 0.0, 0.0, 0.0, 0.0);
}

//Kernel �� ����������
__global__ void	//�����ۼӹ�Լ���� �е��鷳���Ƚ�Block�ڵļ����������ۼӣ�Ȼ��ÿ��Block��ֵ����һ�������Ԫ���
kernel_ZeroOrderJM2H(const hipComplex _coe, const float _k, const cuVector3* _d_p_in, const float* _d_ds, const cuComplexVector3* _d_J_in,
	const cuVector3 _d_p_out, cuComplexVector3* _d_H_out_blocks) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int blockid = int(i / threadsPerBlock);
	//int shift = blockDim.x * blockIdx.x;
	int ii = threadIdx.x;
	
	//��ѡ���ˣ�ֱ��ȫ�㣬���������������ȻΪ0
	__shared__ cuComplexVector3 BlockResults[threadsPerBlock];//���ڴ洢���ڼ�����ɵĽ��
	cuVector3 R = cuVector3Sub(_d_p_out, _d_p_in[i]); //3	//���λ�õ�����λ�õ�ʸ��
	float ds = _d_ds[i];	//����λ�õ����
	cuComplexVector3 J = _d_J_in[i];	//�������
	float absR = cuVector3Abs(R);
	float absR2 = absR*absR;	float absR3 = absR*absR2;
	hipComplex cc = _coe;	//coe should be:
	float kk = _k;

	cuComplexVector3 CV1 = cuComplexVector3Crossfc(R, J);		CV1 = cuComplexVector3Crossfc(R, CV1);
	cuComplexVector3 CV2 = J;
	cuComplexVector3 result;
	hipComplex item1;
	hipComplex item2;
	float kr = -kk*absR;
	hipComplex iteme = make_hipComplex(cos(kr), sin(kr));
	iteme = hipCmulf(cuCmulfcf(iteme, ds), cc);
	item1 = hipCaddf(make_hipComplex(3 - kk*kk*absR2, 0), make_hipComplex(0, -3.0*kr));
	item1 = cuCdivfcf(item1, absR2*absR3);
	item1 = hipCmulf(item1, iteme);
	item2 = hipCaddf(make_hipComplex(float(-1), 0), make_hipComplex(0, kr));
	item2 = cuCmulfcf(item2, float(2.0));
	item2 = hipCmulf(item2, iteme);
	cuComplexVector3AddS(cuComplexVector3Mul(CV1, item1), cuComplexVector3Mul(CV2, item2), result);
	//������ɣ�������Ҫ�����ۼ� �����ڹ����ڴ����ۼ�
	BlockResults[ii] = result;
	__syncthreads();//�������˽���ͬ����һ�����������߳̽���ͬ��

	//if (ii < 128) { BlockResults[ii] = cuComplexVector3Add(BlockResults[ii], BlockResults[ii + 128]); }
	//__syncthreads();//�������˽���ͬ����һ�����������߳̽���ͬ��
	if (ii < 64) { BlockResults[ii] = cuComplexVector3Add(BlockResults[ii], BlockResults[ii + 64]); }
	__syncthreads();//�������˽���ͬ����һ�����������߳̽���ͬ��
	if (ii < 32) { BlockResults[ii] = cuComplexVector3Add(BlockResults[ii], BlockResults[ii + 32]); }
	__syncthreads();//�������˽���ͬ����һ�����������߳̽���ͬ��
	if (ii < 16) { BlockResults[ii] = cuComplexVector3Add(BlockResults[ii], BlockResults[ii + 16]); }
	__syncthreads();//�������˽���ͬ����һ�����������߳̽���ͬ��
	if (ii < 8) { BlockResults[ii] = cuComplexVector3Add(BlockResults[ii], BlockResults[ii + 8]); }
	__syncthreads();//�������˽���ͬ����һ�����������߳̽���ͬ��
	if (ii < 4) { BlockResults[ii] = cuComplexVector3Add(BlockResults[ii], BlockResults[ii + 4]); }
	__syncthreads();//�������˽���ͬ����һ�����������߳̽���ͬ��
	if (ii < 2) { BlockResults[ii] = cuComplexVector3Add(BlockResults[ii], BlockResults[ii + 2]); }
	__syncthreads();//�������˽���ͬ����һ�����������߳̽���ͬ��
	if (ii < 1) {
		BlockResults[ii] = cuComplexVector3Add(BlockResults[ii], BlockResults[ii + 1]);
		_d_H_out_blocks[blockid] = BlockResults[ii];//���
	}
}

__global__ void
kernel_ZeroOrderJ2E(const hipComplex _coe, const float _k, const cuVector3* _d_p_in, const float* _d_ds, const cuComplexVector3* _d_J_in,
	const cuVector3 _d_p_out, cuComplexVector3* _d_E_out_blocks) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int blockid = int(i / threadsPerBlock);
	//int shift = blockDim.x * blockIdx.x;
	int ii = threadIdx.x;

	//��ѡ���ˣ�ֱ��ȫ�㣬���������������ȻΪ0

	__shared__ cuComplexVector3 BlockResults[threadsPerBlock];//���ڴ洢���ڼ�����ɵĽ��
	cuVector3 R = cuVector3Sub(_d_p_out, _d_p_in[i]); //3	//���λ�õ�����λ�õ�ʸ��
	float ds = _d_ds[i];	//����λ�õ����
	cuComplexVector3 J = _d_J_in[i];	//�������
	float absR = cuVector3Abs(R);
	float absR2 = absR*absR;	float absR3 = absR*absR2;
	hipComplex cc = _coe;	//coe should be:
	float kk = _k;

	cuComplexVector3 CV1 = cuComplexVector3Crossfc(R, J);		CV1 = cuComplexVector3Crossfc(R, CV1);
	cuComplexVector3 CV2 = J;
	cuComplexVector3 result;
	hipComplex item1;
	hipComplex item2;
	float kr = -kk*absR;
	hipComplex iteme = make_hipComplex(cos(kr), sin(kr));
	iteme = hipCmulf(cuCmulfcf(iteme, ds), cc);
	item1 = hipCaddf(make_hipComplex(3 - kk*kk*absR2, 0), make_hipComplex(0, -3.0*kr));
	item1 = cuCdivfcf(item1, absR2*absR3);
	item1 = hipCmulf(item1, iteme);
	item2 = hipCaddf(make_hipComplex(float(1), 0), make_hipComplex(0, -kr));
	item2 = cuCmulfcf(item2, float(2.0));
	item2 = hipCmulf(item2, iteme);
	cuComplexVector3AddS(cuComplexVector3Mul(CV1, item1), cuComplexVector3Mul(CV2, item2), result);
	//������ɣ�������Ҫ�����ۼ� �����ڹ����ڴ����ۼ�
	BlockResults[ii] = result;
	__syncthreads();//�������˽���ͬ����һ�����������߳̽���ͬ��

	//if (ii < 128) { BlockResults[ii] = cuComplexVector3Add(BlockResults[ii], BlockResults[ii + 128]); }
	//__syncthreads();//�������˽���ͬ����һ�����������߳̽���ͬ��
	if (ii < 64) { BlockResults[ii] = cuComplexVector3Add(BlockResults[ii], BlockResults[ii + 64]); }
	__syncthreads();//�������˽���ͬ����һ�����������߳̽���ͬ��
	if (ii < 32) { BlockResults[ii] = cuComplexVector3Add(BlockResults[ii], BlockResults[ii + 32]); }
	__syncthreads();//�������˽���ͬ����һ�����������߳̽���ͬ��
	if (ii < 16) { BlockResults[ii] = cuComplexVector3Add(BlockResults[ii], BlockResults[ii + 16]); }
	__syncthreads();//�������˽���ͬ����һ�����������߳̽���ͬ��
	if (ii < 8) { BlockResults[ii] = cuComplexVector3Add(BlockResults[ii], BlockResults[ii + 8]); }
	__syncthreads();//�������˽���ͬ����һ�����������߳̽���ͬ��
	if (ii < 4) { BlockResults[ii] = cuComplexVector3Add(BlockResults[ii], BlockResults[ii + 4]); }
	__syncthreads();//�������˽���ͬ����һ�����������߳̽���ͬ��
	if (ii < 2) { BlockResults[ii] = cuComplexVector3Add(BlockResults[ii], BlockResults[ii + 2]); }
	__syncthreads();//�������˽���ͬ����һ�����������߳̽���ͬ��
	if (ii < 1) {
		BlockResults[ii] = cuComplexVector3Add(BlockResults[ii], BlockResults[ii + 1]);
		_d_E_out_blocks[blockid] = BlockResults[ii];//���
	}
}

int RunJM2H(float _freq,int _NumSource, float* _px_in, float* _py_in, float* _pz_in,
	float* _ds_in, hipComplex* Jmx_in, hipComplex* Jmy_in, hipComplex* Jmz_in,
	int _NumOut, float* _px_out, float* _py_out, float* _pz_out,
	hipComplex* &Hx_out, hipComplex* &Hy_out, hipComplex* &Hz_out) {

	hipDeviceReset;
	hipSetDevice(0);
	FILE* cudalog;
	cudalog = fopen("./cudalog_calculationJM2H.txt", "w");
	fprintf(cudalog, "This is log file for Cuda Calculation \n");
	fclose(cudalog); cudalog = fopen("./cudalog_calculationJM2H.txt", "a");
	hipError_t err = hipSuccess;

	int NumSource = _NumSource;
	//���ٸ�Block
	int blocksPerGrid = (NumSource + threadsPerBlock - 1) / threadsPerBlock;
	//������GPU�ڴ�ʱ�Զ�����
	int NumMalloc = blocksPerGrid*threadsPerBlock;

	//define Host Memory - ����
	cuVector3* h_p_in = nullptr;			h_p_in = new cuVector3[NumSource];
	float* h_ds_in = nullptr;				h_ds_in = new float[NumSource];
	cuComplexVector3* h_J_in = nullptr;		h_J_in = new cuComplexVector3[NumSource];

	for (int i = 0; i < NumSource; i++) {
		h_p_in[i] = SetcuVector3(_px_in[i], _py_in[i], _pz_in[i]);
		h_ds_in[i] = _ds_in[i];
		h_J_in[i] = SetcuComplexVector3c(Jmx_in[i],Jmy_in[i],Jmz_in[i]);
	}

	//define and malloc GPU Memory, consider NumMalloc other than NumSource!!!!
	cuVector3* d_p_in = NULL;		err = hipMalloc((void**)&d_p_in, sizeof(cuVector3)*NumMalloc);
	if (err != hipSuccess) {
		fprintf(cudalog, "Failed to allocate device d_p_in!\n", hipGetErrorString(err));
		fclose(cudalog);
		return EXIT_FAILURE;
	}
	float* d_ds_in = NULL;		err = hipMalloc((void**)&d_ds_in, sizeof(float)*NumMalloc);
	if (err != hipSuccess) {
		fprintf(cudalog, "Failed to allocate device d_ds_in!\n", hipGetErrorString(err));
		fclose(cudalog);
		return EXIT_FAILURE;
	}
	cuComplexVector3* d_J_in = NULL;		err = hipMalloc((void**)&d_J_in, sizeof(cuComplexVector3)*NumMalloc);
	if (err != hipSuccess) {
		fprintf(cudalog, "Failed to allocate device d_J_in!\n", hipGetErrorString(err));
		fclose(cudalog);
		return EXIT_FAILURE;
	}
	fprintf(cudalog, "First, CUDA kernel will launch %d blocks of %d threads for device Input values to Zero on GPU.\n", blocksPerGrid, threadsPerBlock);
	fclose(cudalog); cudalog = fopen("./cudalog_calculationJM2H.txt", "a");
	kernel_SetZero <<< blocksPerGrid, threadsPerBlock >>>
		(d_p_in, d_ds_in, d_J_in);

	//Copy Host Memory into GPU Memory. Notice!! Here should be NumSource
	err = hipMemcpy(d_p_in, h_p_in, sizeof(cuVector3)*NumSource, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		fprintf(cudalog, "Failed to copy memory from host h_p_in to device d_p_in!\n", hipGetErrorString(err));
		fclose(cudalog);
		return EXIT_FAILURE;
	}
	err = hipMemcpy(d_ds_in, h_ds_in, sizeof(float)*NumSource, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		fprintf(cudalog, "Failed to copy memory from host h_ds_in to device d_ds_in!\n", hipGetErrorString(err));
		fclose(cudalog);
		return EXIT_FAILURE;
	}
	err = hipMemcpy(d_J_in, h_J_in, sizeof(cuComplexVector3)*NumSource, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		fprintf(cudalog, "Failed to copy memory from host h_J_in to device d_J_in!\n", hipGetErrorString(err));
		fclose(cudalog);
		return EXIT_FAILURE;
	}

	//Start Physical Optic Computations

	//define CPU and GPU Memory for Buffer storing output results
	//CPU
	cuComplexVector3* h_H_out = nullptr;	h_H_out = new cuComplexVector3[blocksPerGrid];	//���CPU�˵�����
	//GPU
	cuComplexVector3* d_H_out = NULL;		err = hipMalloc((void**)&d_H_out, sizeof(cuComplexVector3)*blocksPerGrid);
	if (err != hipSuccess) {
		fprintf(cudalog, "Failed to allocate device buff d_H_out!\n", hipGetErrorString(err));
		fclose(cudalog);
		return EXIT_FAILURE;
	}
	err = hipMemcpy(h_H_out, d_H_out, sizeof(cuComplexVector3)*blocksPerGrid, hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		fprintf(cudalog, "Failed to copy memory from device d_H_out to host h_H_out,  !\n", hipGetErrorString(err));
		fclose(cudalog);
		return EXIT_FAILURE;
	}

	int NumOut = _NumOut;

	fprintf(cudalog, "Second, CUDA kernel will launch %d blocks of %d threads for Computing Field to Current PO.\n", blocksPerGrid, threadsPerBlock);
	fclose(cudalog); cudalog = fopen("./cudalog_calculationJM2H.txt", "a");

	//computation parameters
	float freq = _freq;
	float lambda = C_Speed / freq;
	float k0 = 2 * Pi / lambda;
	cuVector3 p_out;
	//ע�ⲻͬ����-���ԣ����ֵ��һ��Ŷ
	hipComplex coe;	coe = make_hipComplex(0, 4 * Pi * 2 * Pi*freq*Mu0);
	cuComplexVector3 HResult;
	coe = hipCdivf(make_hipComplex(1.0,0.0),coe);
	for (int i = 0; i < NumOut; i++) {
		p_out = SetcuVector3(_px_out[i], _py_out[i], _pz_out[i]);
		
		kernel_ZeroOrderJM2H <<< blocksPerGrid, threadsPerBlock >>>
			(coe,k0,d_p_in,d_ds_in,d_J_in,p_out,d_H_out);
		//��������ˣ�ÿ��Block����ɼ����һ��cuComplexVector3 �Ļ���ֵ�����䴫�أ����ٸ�Block,���������ж೤��

		err = hipMemcpy(h_H_out, d_H_out, sizeof(cuComplexVector3)*blocksPerGrid, hipMemcpyDeviceToHost);
		if (err != hipSuccess) {
			fprintf(cudalog, "Failed to copy memory from device d_H_out to host h_H_out, at PO iteration step: %d !\n",i, hipGetErrorString(err));
			fclose(cudalog);
			return EXIT_FAILURE;
		}
		//��CPU�˶Է��ص�H��������ۼӣ�GPU�����ۼ�̫���ˣ�
		HResult = SetcuComplexVector3d(0.0, 0.0, 0.0, 0.0, 0.0, 0.0);
		for (int j = 0; j < blocksPerGrid; j++) {
			HResult = cuComplexVector3Add(HResult, h_H_out[j]);
		}
		//�ۼӺ󷵻شų�����
		Hx_out[i] = HResult.x;
		Hy_out[i] = HResult.y;
		Hz_out[i] = HResult.z;

		if(i%1000 == 0){
			fprintf(cudalog, "     CUDA kernel performs %d times of %d for Computing Field to Current PO.\n", i, NumOut, threadsPerBlock);
			fclose(cudalog); cudalog = fopen("./cudalog_calculationJM2H.txt", "a");
		}
	}
	
	//���hipArray
	err = hipFree(d_p_in);	err = hipFree(d_ds_in);	err = hipFree(d_J_in);
	err = hipFree(d_H_out);

	delete[] h_H_out;	h_H_out = nullptr;
	delete[] h_p_in;	h_p_in = nullptr;
	delete[] h_ds_in;	h_ds_in = nullptr;
	delete[] h_J_in;	h_J_in = nullptr;

	return 0;
}

int RunJ2E(float _freq, int _NumSource, float* _px_in, float* _py_in, float* _pz_in,
	float* _ds_in, hipComplex* Jx_in, hipComplex* Jy_in, hipComplex* Jz_in,
	int _NumOut, float* _px_out, float* _py_out, float* _pz_out,
	hipComplex* &Ex_out, hipComplex* &Ey_out, hipComplex* &Ez_out) {

	hipDeviceReset;
	hipSetDevice(0);
	FILE* cudalog;
	cudalog = fopen("./cudalog_calculationJ2E.txt", "w");
	fprintf(cudalog, "This is log file for Cuda Calculation \n");
	fclose(cudalog); cudalog = fopen("./cudalog_calculationJ2E.txt", "a");
	hipError_t err = hipSuccess;

	int NumSource = _NumSource;
	//���ٸ�Block
	int blocksPerGrid = (NumSource + threadsPerBlock - 1) / threadsPerBlock;
	//������GPU�ڴ�ʱ�Զ�����
	int NumMalloc = blocksPerGrid*threadsPerBlock;

	//define Host Memory - ����
	cuVector3* h_p_in = nullptr;			h_p_in = new cuVector3[NumSource];
	float* h_ds_in = nullptr;				h_ds_in = new float[NumSource];
	cuComplexVector3* h_J_in = nullptr;		h_J_in = new cuComplexVector3[NumSource];

	for (int i = 0; i < NumSource; i++) {
		h_p_in[i] = SetcuVector3(_px_in[i], _py_in[i], _pz_in[i]);
		h_ds_in[i] = _ds_in[i];
		h_J_in[i] = SetcuComplexVector3c(Jx_in[i], Jy_in[i], Jz_in[i]);
	}

	//define and malloc GPU Memory, consider NumMalloc other than NumSource!!!!
	cuVector3* d_p_in = NULL;		err = hipMalloc((void**)&d_p_in, sizeof(cuVector3)*NumMalloc);
	if (err != hipSuccess) {
		fprintf(cudalog, "Failed to allocate device d_p_in!\n", hipGetErrorString(err));
		fclose(cudalog);
		return EXIT_FAILURE;
	}
	float* d_ds_in = NULL;		err = hipMalloc((void**)&d_ds_in, sizeof(float)*NumMalloc);
	if (err != hipSuccess) {
		fprintf(cudalog, "Failed to allocate device d_ds_in!\n", hipGetErrorString(err));
		fclose(cudalog);
		return EXIT_FAILURE;
	}
	cuComplexVector3* d_J_in = NULL;		err = hipMalloc((void**)&d_J_in, sizeof(cuComplexVector3)*NumMalloc);
	if (err != hipSuccess) {
		fprintf(cudalog, "Failed to allocate device d_J_in!\n", hipGetErrorString(err));
		fclose(cudalog);
		return EXIT_FAILURE;
	}
	fprintf(cudalog, "First, CUDA kernel will launch %d blocks of %d threads for device Input values to Zero on GPU.\n", blocksPerGrid, threadsPerBlock);
	fclose(cudalog); cudalog = fopen("./cudalog_calculationJ2E.txt", "a");
	kernel_SetZero << < blocksPerGrid, threadsPerBlock >> >
		(d_p_in, d_ds_in, d_J_in);

	//Copy Host Memory into GPU Memory. Notice!! Here should be NumSource
	err = hipMemcpy(d_p_in, h_p_in, sizeof(cuVector3)*NumSource, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		fprintf(cudalog, "Failed to copy memory from host h_p_in to device d_p_in!\n", hipGetErrorString(err));
		fclose(cudalog);
		return EXIT_FAILURE;
	}
	err = hipMemcpy(d_ds_in, h_ds_in, sizeof(float)*NumSource, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		fprintf(cudalog, "Failed to copy memory from host h_ds_in to device d_ds_in!\n", hipGetErrorString(err));
		fclose(cudalog);
		return EXIT_FAILURE;
	}
	err = hipMemcpy(d_J_in, h_J_in, sizeof(cuComplexVector3)*NumSource, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		fprintf(cudalog, "Failed to copy memory from host h_J_in to device d_J_in!\n", hipGetErrorString(err));
		fclose(cudalog);
		return EXIT_FAILURE;
	}

	//Start Physical Optic Computations

	//define CPU and GPU Memory for Buffer storing output results
	//CPU
	cuComplexVector3* h_E_out = nullptr;	h_E_out = new cuComplexVector3[blocksPerGrid];
	//GPU
	cuComplexVector3* d_E_out = NULL;		err = hipMalloc((void**)&d_E_out, sizeof(cuComplexVector3)*blocksPerGrid);
	if (err != hipSuccess) {
		fprintf(cudalog, "Failed to allocate device buff d_H_out!\n", hipGetErrorString(err));
		fclose(cudalog);
		return EXIT_FAILURE;
	}
	int NumOut = _NumOut;

	fprintf(cudalog, "Second, CUDA kernel will launch %d blocks of %d threads for Computing Field to Current PO.\n", blocksPerGrid, threadsPerBlock);
	fclose(cudalog); cudalog = fopen("./cudalog_calculationJ2E.txt", "a");

	//computation parameters
	float freq = _freq;
	float lambda = C_Speed / freq;
	float k0 = 2 * Pi / lambda;
	cuVector3 p_out;
	//ע�ⲻͬ����-���ԣ����ֵ��һ��Ŷ
	hipComplex coe;	coe = make_hipComplex(0, 4 * Pi * 2 * Pi*freq*Eps0);
	cuComplexVector3 EResult;
	coe = hipCdivf(make_hipComplex(1.0, 0.0), coe);
	for (int i = 0; i < NumOut; i++) {
		p_out = SetcuVector3(_px_out[i], _py_out[i], _pz_out[i]);

		kernel_ZeroOrderJ2E << < blocksPerGrid, threadsPerBlock >> >
			(coe, k0, d_p_in, d_ds_in, d_J_in, p_out, d_E_out);
		//��������ˣ�ÿ��Block����ɼ����һ��cuComplexVector3 �Ļ���ֵ�����䴫�أ����ٸ�Block,���������ж೤��
		err = hipMemcpy(h_E_out, d_E_out, sizeof(cuComplexVector3)*blocksPerGrid, hipMemcpyDeviceToHost);
		if (err != hipSuccess) {
			fprintf(cudalog, "Failed to copy memory from device d_H_out to host h_H_out, at PO iteration step: %d !\n", i, hipGetErrorString(err));
			fclose(cudalog);
			return EXIT_FAILURE;
		}
		//��CPU�˶Է��ص�H��������ۼӣ�GPU�����ۼ�̫���ˣ�
		EResult = SetcuComplexVector3d(0.0, 0.0, 0.0, 0.0, 0.0, 0.0);
		for (int j = 0; j < blocksPerGrid; j++) {
			EResult = cuComplexVector3Add(EResult, h_E_out[j]);
		}
		//�ۼӺ󷵻شų�����
		Ex_out[i] = EResult.x;
		Ey_out[i] = EResult.y;
		Ez_out[i] = EResult.z;

		if (i % 1000 == 0) {
			fprintf(cudalog, "     CUDA kernel performs %d times of %d for Computing Current to Field PO.\n", i, NumOut, threadsPerBlock);
			fclose(cudalog); cudalog = fopen("./cudalog_calculationJ2E.txt", "a");
		}
	}

	//���hipArray
	err = hipFree(d_p_in);	err = hipFree(d_ds_in);	err = hipFree(d_J_in);
	err = hipFree(d_E_out);

	delete[] h_E_out;	h_E_out = nullptr;
	delete[] h_p_in;	h_p_in = nullptr;
	delete[] h_ds_in;	h_ds_in = nullptr;
	delete[] h_J_in;	h_J_in = nullptr;

	return 0;
}
