#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "./hip/hip_runtime_api.h"
#include ""
#include "hip/device_functions.h"
#include "CUDADLL.h"
#include "stdio.h"
#include "stdlib.h"
#include "thrust/device_vector.h"
#include "thrust/host_vector.h"
using namespace thrust;

const int threadsPerBlock = 64;

#if CUDART_VERSION < 5000

// CUDA-C includes
#include <hip/hip_runtime.h>


// This function wraps the CUDA Driver API into a template function
#include
template <class T>
inline void getCudaAttribute(T *attribute, hipDeviceAttribute_t device_attribute, int device)
{
	hipError_t error = hipDeviceGetAttribute(attribute, device_attribute, device);

	if (hipSuccess != error)
	{
		fprintf(stderr, "cuSafeCallNoSync() Driver API error = %04d from file <%s>, line %i.\n",
			error, __FILE__, __LINE__);

		exit(EXIT_FAILURE);
	}
}

#endif /* CUDART_VERSION < 5000 */

//Total Shared 
//Shared 49152 bytes
__device__ //������Դ�Device ��
inline void Vector3Add(const float x1, const float y1, const float z1, const float x2, const float y2, const float z2, float & x3, float & y3, float & z3) {
	x3 = x1 + x2;
	y3 = y1 + y2;
	z3 = z1 + z2;
}
__device__
inline void Vector3Sub(const float x1, const float y1, const float z1, const float x2, const float y2, const float z2, float & x3, float & y3, float & z3) {
	x3 = x1 - x2;
	y3 = y1 - y2;
	z3 = z1 - z2;
}
__device__
inline void Vector3Dot(const float x1, const float y1, const float z1, const float x2, const float y2, const float z2, float & dotresult) {
	dotresult = x1*x2 + y1*y2 + z1*z2;
}
__device__
inline void Vector3Cross(const float x1, const float y1, const float z1, const float x2, const float y2, const float z2, float & x3, float & y3, float & z3) {
	x3 = y1*z2 - z1*y2;
	y3 = z1*x2 - x1*z2;
	z3 = x1*y2 - y1*x2;

	//y * v.z - z * v.y,
	//z * v.x - x * v.z,
	//x * v.y - y * v.x
}

__global__ void
FindInterSection_register(const float _psx, const float _psy, const float _psz, const float _dsx, const float _dsy, const float _dsz,
	const int _sourceIndex,
	const float* f_stlp1x, const float* f_stlp1y, const float* f_stlp1z,
	const float* f_stlp2x, const float* f_stlp2y, const float* f_stlp2z,
	const float* f_stlp3x, const float* f_stlp3y, const float* f_stlp3z,
	bool* f_intersecb, float* f_prot, int* f_STLIndex,
	float* f_inter_x, float* f_inter_y, float* f_inter_z,
	const int numElements)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	//int shift = blockDim.x * blockIdx.x;
	int ii = threadIdx.x;

	if (i < numElements)
	{

		//C[i] = A[i] + B[i];
		float s_stlp1x = f_stlp1x[i];		float s_stlp2x = f_stlp2x[i];		float s_stlp3x = f_stlp3x[i];
		float s_stlp1y = f_stlp1y[i];		float s_stlp2y = f_stlp2y[i];		float s_stlp3y = f_stlp3y[i];
		float s_stlp1z = f_stlp1z[i];		float s_stlp2z = f_stlp2z[i];		float s_stlp3z = f_stlp3z[i];

		float s_psx = _psx;	float s_psy = _psy;	float s_psz = _psz;
		float s_dsx = _dsx;	float s_dsy = _dsy;	float s_dsz = _dsz;
		float E1_x = 0;	float E1_y = 0;	float E1_z = 0;
		float E2_x = 0; float E2_y = 0; float E2_z = 0;
		float P_x = 0;	float P_y = 0;	float P_z = 0;
		float Q_x = 0;	float Q_y = 0;	float Q_z = 0;
		float T_x = 0;	float T_y = 0;	float T_z = 0;
		float u = 0;	float v = 0;	float det = 0;
		float s_prot = 0;
		bool s_intersected = true;
		int s_STLIndex = 0;
		float inter_x = 0;
		float inter_y = 0;
		float inter_z = 0;

		Vector3Sub(s_stlp2x, s_stlp2y, s_stlp2z, s_stlp1x, s_stlp1y, s_stlp1z, E1_x, E1_y, E1_z);
		Vector3Sub(s_stlp3x, s_stlp3y, s_stlp3z, s_stlp1x, s_stlp1y, s_stlp1z, E2_x, E2_y, E2_z);
		Vector3Cross(s_dsx, s_dsy, s_dsz, E2_x, E2_y, E2_z, P_x, P_y, P_z);
		Vector3Dot(E1_x, E1_y, E1_z, P_x, P_y, P_z, det);

		Vector3Sub(s_psx, s_psy, s_psz, s_stlp1x, s_stlp1y, s_stlp1z, T_x, T_y, T_z);
		Vector3Dot(T_x, T_y, T_z, P_x, P_y, P_z, u);
		u = u / det;

		Vector3Cross(T_x, T_y, T_z, E1_x, E1_y, E1_z, Q_x, Q_y, Q_z);
		Vector3Dot(s_dsx, s_dsy, s_dsz, Q_x, Q_y, Q_z, v);
		v = v / det;

		__syncthreads();

		if (u < 0.0 || u>1 || v < 0 || u + v>1) {
			//û�н���
		}
		else {
			

			//�н���
			Vector3Dot(E2_x, E2_y, E2_z, Q_x, Q_y, Q_z, s_prot);
			s_prot = s_prot / det;
			Vector3Add(s_psx, s_psy, s_psz, s_dsx*s_prot, s_dsy*s_prot, s_dsz*s_prot, inter_x, inter_y, inter_z);
			s_intersected = true;
			s_STLIndex = i;
			//��������ص�ȫ���ڴ�
			f_STLIndex[_sourceIndex] = s_STLIndex;
			f_intersecb[_sourceIndex] = s_intersected;
			f_prot[_sourceIndex] = s_prot;
			f_inter_x[_sourceIndex] = inter_x;
			f_inter_y[_sourceIndex] = inter_y;
			f_inter_z[_sourceIndex] = inter_z;
			//atomic_store(&s_intersected,&f_intersecb[_sourceIndex]);
			//atomic_store(&s_STLIndex, &f_STLIndex[_sourceIndex]);
			
		}//else

	}
}


/*
//���빲���ڴ�
__shared__ float s_stlp1x[threadsPerBlock];	__shared__ float s_stlp1y[threadsPerBlock];	__shared__ float s_stlp1z[threadsPerBlock];
__shared__ float s_stlp2x[threadsPerBlock];	__shared__ float s_stlp2y[threadsPerBlock];	__shared__ float s_stlp2z[threadsPerBlock];
__shared__ float s_stlp3x[threadsPerBlock];	__shared__ float s_stlp3y[threadsPerBlock];	__shared__ float s_stlp3z[threadsPerBlock];
__shared__ float E1_x[threadsPerBlock];
__shared__ float E1_y[threadsPerBlock];
__shared__ float E1_z[threadsPerBlock];
__shared__ float E2_x[threadsPerBlock];
__shared__ float E2_y[threadsPerBlock];
__shared__ float E2_z[threadsPerBlock];
__shared__ float P_x[threadsPerBlock];
__shared__ float P_y[threadsPerBlock];
__shared__ float P_z[threadsPerBlock];
__shared__ float Q_x[threadsPerBlock];
__shared__ float Q_y[threadsPerBlock];
__shared__ float Q_z[threadsPerBlock];
__shared__ float det[threadsPerBlock];
__shared__ float u[threadsPerBlock];
__shared__ float v[threadsPerBlock];
__shared__ float T_x[threadsPerBlock];
__shared__ float T_y[threadsPerBlock];
__shared__ float T_z[threadsPerBlock];
__shared__ float s_psx;	__shared__ float s_psy;	__shared__ float s_psz;
__shared__ float s_dsx;	__shared__ float s_dsy;	__shared__ float s_dsz;
//__shared__ int s_interInt;
__shared__ int s_STLIndex;
__shared__ float s_prot;
__shared__ float inter_x;
__shared__ float inter_y;
__shared__ float inter_z;
__shared__ bool s_intersected;

__global__ void
FindInterSection(const float _psx, const float _psy, const float _psz, const float _dsx, const float _dsy, const float _dsz,
	             const int _sourceIndex,
				 const float* f_stlp1x, const float* f_stlp1y, const float* f_stlp1z,
				 const float* f_stlp2x, const float* f_stlp2y, const float* f_stlp2z, 
				 const float* f_stlp3x, const float* f_stlp3y, const float* f_stlp3z, 
				 bool* f_intersecb, float* f_prot, int* f_STLIndex, 
				 float* f_inter_x, float* f_inter_y, float* f_inter_z,
			     const int numElements)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int shift = blockDim.x * blockIdx.x;
	int ii = threadIdx.x;

	if (i < numElements)
	{

		//C[i] = A[i] + B[i];
		s_stlp1x[ii] = f_stlp1x[i];		s_stlp2x[ii] = f_stlp2x[i];		s_stlp3x[ii] = f_stlp3x[i];
		s_stlp1y[ii] = f_stlp1y[i];		s_stlp2y[ii] = f_stlp2y[i];		s_stlp3y[ii] = f_stlp3y[i];
		s_stlp1z[ii] = f_stlp1z[i];		s_stlp2z[ii] = f_stlp2z[i];		s_stlp3z[ii] = f_stlp3z[i];
		
		s_psx = _psx;	s_psy = _psy;	s_psz = _psz;
		s_dsx = _dsx;	s_dsy = _dsy;	s_dsz = _dsz;
		
		Vector3Sub(s_stlp2x[ii], s_stlp2y[ii], s_stlp2z[ii], s_stlp1x[ii], s_stlp1y[ii], s_stlp1z[ii], E1_x[ii], E1_y[ii], E1_z[ii]);
		Vector3Sub(s_stlp3x[ii], s_stlp3y[ii], s_stlp3z[ii], s_stlp1x[ii], s_stlp1y[ii], s_stlp1z[ii], E2_x[ii], E2_y[ii], E2_z[ii]);
		Vector3Cross(s_dsx, s_dsy, s_dsz, E2_x[ii], E2_y[ii], E2_z[ii], P_x[ii], P_y[ii], P_z[ii]);
		Vector3Dot(E1_x[ii], E1_y[ii], E1_z[ii], P_x[ii], P_y[ii], P_z[ii], det[ii]);
		Vector3Sub(s_dsx,s_dsy,s_dsz,s_stlp1x[ii], s_stlp1y[ii], s_stlp1z[ii],T_x[ii],T_y[ii],T_z[ii]);
		Vector3Dot(T_x[ii], T_y[ii], T_z[ii], P_x[ii], P_y[ii], P_z[ii], u[ii]);
		u[ii] = u[ii] / det[ii];

		Vector3Cross(T_x[ii], T_y[ii], T_z[ii], E1_x[ii], E1_y[ii], E1_z[ii], Q_x[ii], Q_y[ii], Q_z[ii]);
		Vector3Dot(s_dsx, s_dsy, s_dsz, Q_x[ii], Q_y[ii], Q_z[ii], v[ii]);
		v[ii] = v[ii] / det[ii];
		
		__syncthreads();
		
		if (u[ii] < 0.0 || u[ii]>1 || v[ii] < 0 || u[ii] + v[ii]>1) {
			//û�н���
		}
		else {
			//�н���
			Vector3Dot(E2_x[ii], E2_y[ii], E2_z[ii], Q_x[ii], Q_y[ii], Q_z[ii], s_prot);
			s_prot = s_prot / det[ii];
			Vector3Add(s_psx,s_psy,s_psz,s_dsx*s_prot,s_dsy*s_prot,s_dsz*s_prot,inter_x,inter_y,inter_z);
			s_intersected = true;
			s_STLIndex = i;
			//��������ص�ȫ���ڴ�
			f_STLIndex[_sourceIndex] = s_STLIndex;
			f_intersecb[_sourceIndex] = s_intersected;
			f_prot[_sourceIndex] = s_prot;
			f_inter_x[_sourceIndex] = inter_x;
			f_inter_y[_sourceIndex] = inter_y;
			f_inter_z[_sourceIndex] = inter_z;
			//atomic_store(&s_intersected,&f_intersecb[_sourceIndex]);
			//atomic_store(&s_STLIndex, &f_STLIndex[_sourceIndex]);

		}//else
		
	}
}
*/

__global__  void
SetResultsZero(const int f_NumSource, const int f_NumSTL, bool* f_intersecb, float* f_prot, int* f_STLIndex, float* f_inter_x, float* f_inter_y, float* f_inter_z) 
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < f_NumSource) {
		f_intersecb[i] = false;
		f_prot[i] = 0;
		f_STLIndex[i] = f_NumSTL + 1;
		f_inter_x[i] = 0.0;
		f_inter_y[i] = 0.0;
		f_inter_z[i] = 0.0;
	}
}





void DeviceInf(void) {
	FILE* cudalog;
	cudalog = fopen("./cudalog.txt", "w");
	fprintf(cudalog," CUDA Device Query (Runtime API) version (CUDART static linking)\n\n");
	//��ѯ�м���CUDA Device
	int deviceCount = 0;
	hipError_t error_id = hipGetDeviceCount(&deviceCount);
	if (error_id != hipSuccess)
	{
		fprintf(cudalog,"hipGetDeviceCount returned %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id));
		fprintf(cudalog,"Result = FAIL\n");
		exit(EXIT_FAILURE);
	}
	//��ʾ�м���CUDA Device
	// This function call returns 0 if there are no CUDA capable devices.
	if (deviceCount == 0)
	{
		fprintf(cudalog,"There are no available device(s) that support CUDA\n");
	}
	else
	{
		fprintf(cudalog,"Detected %d CUDA Capable device(s)\n", deviceCount);
	}
	//���β�ѯÿ��Device
	int dev, driverVersion = 0, runtimeVersion = 0;
	for (dev = 0; dev < deviceCount; ++dev)
	{
		hipSetDevice(dev);
		hipDeviceProp_t deviceProp;
		hipGetDeviceProperties(&deviceProp, dev);

		printf("\nDevice %d: \"%s\"\n", dev, deviceProp.name);

		// Console log
		hipDriverGetVersion(&driverVersion);
		hipRuntimeGetVersion(&runtimeVersion);
		fprintf(cudalog,"  CUDA Driver Version / Runtime Version          %d.%d / %d.%d\n", driverVersion / 1000, (driverVersion % 100) / 10, runtimeVersion / 1000, (runtimeVersion % 100) / 10);
		fprintf(cudalog,"  CUDA Capability Major/Minor version number:    %d.%d\n", deviceProp.major, deviceProp.minor);

		char msg[256];
		SPRINTF(msg, "  Total amount of global memory:                 %.0f MBytes (%llu bytes)\n",
			(float)deviceProp.totalGlobalMem / 1048576.0f, (unsigned long long) deviceProp.totalGlobalMem);
		fprintf(cudalog,"%s", msg);

		fprintf(cudalog,"  (%2d) Multiprocessors, (%3d) CUDA Cores/MP:     %d CUDA Cores\n",
			deviceProp.multiProcessorCount,
			_ConvertSMVer2Cores(deviceProp.major, deviceProp.minor),
			_ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) * deviceProp.multiProcessorCount);
		fprintf(cudalog,"  GPU Max Clock rate:                            %.0f MHz (%0.2f GHz)\n", deviceProp.clockRate * 1e-3f, deviceProp.clockRate * 1e-6f);


#if CUDART_VERSION >= 5000
		// This is supported in CUDA 5.0 (runtime API device properties)
		fprintf(cudalog,"  Memory Clock rate:                             %.0f Mhz\n", deviceProp.memoryClockRate * 1e-3f);
		fprintf(cudalog,"  Memory Bus Width:                              %d-bit\n", deviceProp.memoryBusWidth);

		if (deviceProp.l2CacheSize)
		{
			fprintf(cudalog,"  L2 Cache Size:                                 %d bytes\n", deviceProp.l2CacheSize);
		}

#else
		// This only available in CUDA 4.0-4.2 (but these were only exposed in the CUDA Driver API)
		int memoryClock;
		getCudaAttribute<int>(&memoryClock, hipDeviceAttributeMemoryClockRate, dev);
		fprintf(cudalog,"  Memory Clock rate:                             %.0f Mhz\n", memoryClock * 1e-3f);
		int memBusWidth;
		getCudaAttribute<int>(&memBusWidth, hipDeviceAttributeMemoryBusWidth, dev);
		fprintf(cudalog,"  Memory Bus Width:                              %d-bit\n", memBusWidth);
		int L2CacheSize;
		getCudaAttribute<int>(&L2CacheSize, hipDeviceAttributeL2CacheSize, dev);

		if (L2CacheSize)
		{
			fprintf(cudalog,"  L2 Cache Size:                                 %d bytes\n", L2CacheSize);
		}

#endif

		fprintf(cudalog,"  Maximum Texture Dimension Size (x,y,z)         1D=(%d), 2D=(%d, %d), 3D=(%d, %d, %d)\n",
			deviceProp.maxTexture1D, deviceProp.maxTexture2D[0], deviceProp.maxTexture2D[1],
			deviceProp.maxTexture3D[0], deviceProp.maxTexture3D[1], deviceProp.maxTexture3D[2]);
		fprintf(cudalog,"  Maximum Layered 1D Texture Size, (num) layers  1D=(%d), %d layers\n",
			deviceProp.maxTexture1DLayered[0], deviceProp.maxTexture1DLayered[1]);
		fprintf(cudalog,"  Maximum Layered 2D Texture Size, (num) layers  2D=(%d, %d), %d layers\n",
			deviceProp.maxTexture2DLayered[0], deviceProp.maxTexture2DLayered[1], deviceProp.maxTexture2DLayered[2]);


		fprintf(cudalog,"  Total amount of constant memory:               %lu bytes\n", deviceProp.totalConstMem);
		fprintf(cudalog, "  Total amount of shared memory per block:       %lu bytes\n", deviceProp.sharedMemPerBlock);
		fprintf(cudalog, "  Total number of registers available per block: %d\n", deviceProp.regsPerBlock);
		fprintf(cudalog, "  Warp size:                                     %d\n", deviceProp.warpSize);
		fprintf(cudalog, "  Maximum number of threads per multiprocessor:  %d\n", deviceProp.maxThreadsPerMultiProcessor);
		fprintf(cudalog, "  Maximum number of threads per block:           %d\n", deviceProp.maxThreadsPerBlock);
		fprintf(cudalog, "  Max dimension size of a thread block (x,y,z): (%d, %d, %d)\n",
			deviceProp.maxThreadsDim[0],
			deviceProp.maxThreadsDim[1],
			deviceProp.maxThreadsDim[2]);
		fprintf(cudalog, "  Max dimension size of a grid size    (x,y,z): (%d, %d, %d)\n",
			deviceProp.maxGridSize[0],
			deviceProp.maxGridSize[1],
			deviceProp.maxGridSize[2]);
		fprintf(cudalog, "  Maximum memory pitch:                          %lu bytes\n", deviceProp.memPitch);
		fprintf(cudalog, "  Texture alignment:                             %lu bytes\n", deviceProp.textureAlignment);
		fprintf(cudalog, "  Concurrent copy and kernel execution:          %s with %d copy engine(s)\n", (deviceProp.deviceOverlap ? "Yes" : "No"), deviceProp.asyncEngineCount);
		fprintf(cudalog, "  Run time limit on kernels:                     %s\n", deviceProp.kernelExecTimeoutEnabled ? "Yes" : "No");
		fprintf(cudalog, "  Integrated GPU sharing Host Memory:            %s\n", deviceProp.integrated ? "Yes" : "No");
		fprintf(cudalog, "  Support host page-locked memory mapping:       %s\n", deviceProp.canMapHostMemory ? "Yes" : "No");
		fprintf(cudalog, "  Alignment requirement for Surfaces:            %s\n", deviceProp.surfaceAlignment ? "Yes" : "No");
		fprintf(cudalog, "  Device has ECC support:                        %s\n", deviceProp.ECCEnabled ? "Enabled" : "Disabled");
#if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
		fprintf(cudalog,"  CUDA Device Driver Mode (TCC or WDDM):         %s\n", deviceProp.tccDriver ? "TCC (Tesla Compute Cluster Driver)" : "WDDM (Windows Display Driver Model)");
#endif
		fprintf(cudalog,"  Device supports Unified Addressing (UVA):      %s\n", deviceProp.unifiedAddressing ? "Yes" : "No");
		fprintf(cudalog,"  Device PCI Domain ID / Bus ID / location ID:   %d / %d / %d\n", deviceProp.pciDomainID, deviceProp.pciBusID, deviceProp.pciDeviceID);

		const char *sComputeMode[] =
		{
			"Default (multiple host threads can use ::hipSetDevice() with device simultaneously)",
			"Exclusive (only one host thread in one process is able to use ::hipSetDevice() with this device)",
			"Prohibited (no host thread can use ::hipSetDevice() with this device)",
			"Exclusive Process (many threads in one process is able to use ::hipSetDevice() with this device)",
			"Unknown",
			NULL
		};
		fprintf(cudalog,"  Compute Mode:\n");
		fprintf(cudalog,"     < %s >\n", sComputeMode[deviceProp.computeMode]);
	}

	// If there are 2 or more GPUs, query to determine whether RDMA is supported
	if (deviceCount >= 2)
	{
		hipDeviceProp_t prop[64];
		int gpuid[64]; // we want to find the first two GPUs that can support P2P
		int gpu_p2p_count = 0;

		for (int i = 0; i < deviceCount; i++)
		{
			checkCudaErrors(hipGetDeviceProperties(&prop[i], i));

			// Only boards based on Fermi or later can support P2P
			if ((prop[i].major >= 2)
#if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
				// on Windows (64-bit), the Tesla Compute Cluster driver for windows must be enabled to support this
				&& prop[i].tccDriver
#endif
				)
			{
				// This is an array of P2P capable GPUs
				gpuid[gpu_p2p_count++] = i;
			}
		}

		// Show all the combinations of support P2P GPUs
		int can_access_peer;

		if (gpu_p2p_count >= 2)
		{
			for (int i = 0; i < gpu_p2p_count; i++)
			{
				for (int j = 0; j < gpu_p2p_count; j++)
				{
					if (gpuid[i] == gpuid[j])
					{
						continue;
					}
					checkCudaErrors(hipDeviceCanAccessPeer(&can_access_peer, gpuid[i], gpuid[j]));
					fprintf(cudalog,"> Peer access from %s (GPU%d) -> %s (GPU%d) : %s\n", prop[gpuid[i]].name, gpuid[i],
						prop[gpuid[j]].name, gpuid[j],
						can_access_peer ? "Yes" : "No");
				}
			}
		}
	}

	// csv masterlog info
	// *****************************
	// exe and CUDA driver name
	fprintf(cudalog,"\n");
	std::string sProfileString = "deviceQuery, CUDA Driver = CUDART";
	char cTemp[16];

	// driver version
	sProfileString += ", CUDA Driver Version = ";
#if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
	sprintf_s(cTemp, 10, "%d.%d", driverVersion / 1000, (driverVersion % 100) / 10);
#else
	sprintf(cTemp, "%d.%d", driverVersion / 1000, (driverVersion % 100) / 10);
#endif
	sProfileString += cTemp;

	// Runtime version
	sProfileString += ", CUDA Runtime Version = ";
#if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
	sprintf_s(cTemp, 10, "%d.%d", runtimeVersion / 1000, (runtimeVersion % 100) / 10);
#else
	sprintf(cTemp, "%d.%d", runtimeVersion / 1000, (runtimeVersion % 100) / 10);
#endif
	sProfileString += cTemp;

	// Device count
	sProfileString += ", NumDevs = ";
#if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
	sprintf_s(cTemp, 10, "%d", deviceCount);
#else
	sprintf(cTemp, "%d", deviceCount);
#endif
	sProfileString += cTemp;

	// Print Out all device Names
	for (dev = 0; dev < deviceCount; ++dev)
	{
#if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
		sprintf_s(cTemp, 13, ", Device%d = ", dev);
#else
		sprintf(cTemp, ", Device%d = ", dev);
#endif
		hipDeviceProp_t deviceProp;
		hipGetDeviceProperties(&deviceProp, dev);
		sProfileString += cTemp;
		sProfileString += deviceProp.name;
	}

	sProfileString += "\n";
	fprintf(cudalog, "%s", sProfileString.c_str());

	fprintf(cudalog, "Result = PASS\n");


	fclose(cudalog);


}

void RunReflectionLine(int _NumPoints, float* _psourcex, float* _psourcey, float*  _psourcez,
											  float* _pdirx, float*  _pdiry, float*  _pdirz,
											  bool* &_intersected, float* &_prot,int* &_STLIndex,
												float* &_inter_x, float* &_inter_y, float* &_inter_z,
												int _NumSTL, float* _stlp1x, float* _stlp1y, float* _stlp1z,
												float* _stlp2x, float* _stlp2y, float* _stlp2z,
												float* _stlp3x, float* _stlp3y, float* _stlp3z) {
	hipDeviceReset;
	hipSetDevice(0);
	FILE* cudalog;
	cudalog = fopen("./cudalog_calculation.txt", "w");
	fprintf(cudalog, "This is log file for Cuda Calculation \n");
	fclose(cudalog); cudalog = fopen("./cudalog_calculation.txt", "a");
	hipError_t err = hipSuccess;
	int NumSTL = _NumSTL;

	//Դ�ĳߴ�
	size_t sizeSource = _NumPoints * sizeof(float);
	size_t sizeSourceBool = _NumPoints * sizeof(bool);
	size_t sizeSourceInt = _NumPoints * sizeof(int);
	//����ĳߴ�
	size_t sizeSTL = _NumSTL * sizeof(float);
	//����GPU�ڴ� - ������
	//����Դ
	float* d_psourcex = NULL;		err = hipMalloc((void **)&d_psourcex, sizeSource);
	if (err != hipSuccess) {
		fprintf(cudalog, "Failed to allocate device d_psourcex!\n", hipGetErrorString(err));
		fclose(cudalog);
		exit(EXIT_FAILURE);
	}
	err = hipMemcpy(d_psourcex, _psourcex, sizeSource, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		fprintf(cudalog, "Failed to copy memory from host d_psourcex!\n", hipGetErrorString(err));
		fclose(cudalog);
		exit(EXIT_FAILURE);
	}

	float* d_psourcey = NULL;		err = hipMalloc((void **)&d_psourcey, sizeSource);
	if (err != hipSuccess) {
		fprintf(cudalog, "Failed to allocate device d_psourcey!\n", hipGetErrorString(err));
		fclose(cudalog);
		exit(EXIT_FAILURE);
	}
	err = hipMemcpy(d_psourcey, _psourcey, sizeSource, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		fprintf(cudalog, "Failed to copy memory from host d_psourcey!\n", hipGetErrorString(err));
		fclose(cudalog);
		exit(EXIT_FAILURE);
	}

	float* d_psourcez = NULL;		err = hipMalloc((void **)&d_psourcez, sizeSource);
	if (err != hipSuccess) {
		fprintf(cudalog, "Failed to allocate device d_psourcez!\n", hipGetErrorString(err));
		fclose(cudalog);
		exit(EXIT_FAILURE);
	}
	err = hipMemcpy(d_psourcez, _psourcez, sizeSource, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		fprintf(cudalog, "Failed to copy memory from host d_psourcez!\n", hipGetErrorString(err));
		fclose(cudalog);
		exit(EXIT_FAILURE);
	}

	float* d_pdirx = NULL;			err = hipMalloc((void **)&d_pdirx, sizeSource);
	if (err != hipSuccess) {
		fprintf(cudalog, "Failed to allocate device d_pdirx!\n", hipGetErrorString(err));
		fclose(cudalog);
		exit(EXIT_FAILURE);
	}
	err = hipMemcpy(d_pdirx, _pdirx, sizeSource, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		fprintf(cudalog, "Failed to copy memory from host d_pdirx!\n", hipGetErrorString(err));
		fclose(cudalog);
		exit(EXIT_FAILURE);
	}

	float* d_pdiry = NULL;			err = hipMalloc((void **)&d_pdiry, sizeSource);
	if (err != hipSuccess) {
		fprintf(cudalog, "Failed to allocate device d_pdiry!\n", hipGetErrorString(err));
		fclose(cudalog);
		exit(EXIT_FAILURE);
	}
	err = hipMemcpy(d_pdiry, _pdiry, sizeSource, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		fprintf(cudalog, "Failed to copy memory from host d_pdiry!\n", hipGetErrorString(err));
		fclose(cudalog);
		exit(EXIT_FAILURE);
	}

	float* d_pdirz = NULL;			err = hipMalloc((void **)&d_pdirz, sizeSource);
	if (err != hipSuccess) {
		fprintf(cudalog, "Failed to allocate device d_pdirz!\n", hipGetErrorString(err));
		fclose(cudalog);
		exit(EXIT_FAILURE);
	}
	err = hipMemcpy(d_pdirz, _pdirz, sizeSource, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		fprintf(cudalog, "Failed to copy memory from host d_pdirz!\n", hipGetErrorString(err));
		fclose(cudalog);
		exit(EXIT_FAILURE);
	}
	
	//�������� 3���������
	//��1
	float* d_stlp1x = NULL;			err = hipMalloc((void **)&d_stlp1x, sizeSTL);
	if (err != hipSuccess) {
		fprintf(cudalog, "Failed to allocate device d_stlp1x!\n", hipGetErrorString(err));
		fclose(cudalog);
		exit(EXIT_FAILURE);
	}
	err = hipMemcpy(d_stlp1x, _stlp1x, sizeSTL, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		fprintf(cudalog, "Failed to copy memory from host d_stlp1x!\n", hipGetErrorString(err));
		fclose(cudalog);
		exit(EXIT_FAILURE);
	}

	float* d_stlp1y = NULL;			err = hipMalloc((void **)&d_stlp1y, sizeSTL);
	if (err != hipSuccess) {
		fprintf(cudalog, "Failed to allocate device d_stlp1y!\n", hipGetErrorString(err));
		fclose(cudalog);
		exit(EXIT_FAILURE);
	}
	err = hipMemcpy(d_stlp1y, _stlp1y, sizeSTL, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		fprintf(cudalog, "Failed to copy memory from host d_stlp1y!\n", hipGetErrorString(err));
		fclose(cudalog);
		exit(EXIT_FAILURE);
	}

	float* d_stlp1z = NULL;			err = hipMalloc((void **)&d_stlp1z, sizeSTL);
	if (err != hipSuccess) {
		fprintf(cudalog, "Failed to allocate device d_stlp1z!\n", hipGetErrorString(err));
		fclose(cudalog);
		exit(EXIT_FAILURE);
	}
	err = hipMemcpy(d_stlp1z, _stlp1z, sizeSTL, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		fprintf(cudalog, "Failed to copy memory from host d_stlp1z!\n", hipGetErrorString(err));
		fclose(cudalog);
		exit(EXIT_FAILURE);
	}

	//��2
	float* d_stlp2x = NULL;			err = hipMalloc((void **)&d_stlp2x, sizeSTL);
	if (err != hipSuccess) {
		fprintf(cudalog, "Failed to allocate device d_stlp2x!\n", hipGetErrorString(err));
		fclose(cudalog);
		exit(EXIT_FAILURE);
	}
	err = hipMemcpy(d_stlp2x, _stlp2x, sizeSTL, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		fprintf(cudalog, "Failed to copy memory from host d_stlp2x!\n", hipGetErrorString(err));
		fclose(cudalog);
		exit(EXIT_FAILURE);
	}

	float* d_stlp2y = NULL;			err = hipMalloc((void **)&d_stlp2y, sizeSTL);
	if (err != hipSuccess) {
		fprintf(cudalog, "Failed to allocate device d_stlp2y!\n", hipGetErrorString(err));
		fclose(cudalog);
		exit(EXIT_FAILURE);
	}
	err = hipMemcpy(d_stlp2y, _stlp2y, sizeSTL, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		fprintf(cudalog, "Failed to copy memory from host d_stlp2y!\n", hipGetErrorString(err));
		fclose(cudalog);
		exit(EXIT_FAILURE);
	}

	float* d_stlp2z = NULL;			err = hipMalloc((void **)&d_stlp2z, sizeSTL);
	if (err != hipSuccess) {
		fprintf(cudalog, "Failed to allocate device d_stlp2z!\n", hipGetErrorString(err));
		fclose(cudalog);
		exit(EXIT_FAILURE);
	}
	err = hipMemcpy(d_stlp2z, _stlp2z, sizeSTL, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		fprintf(cudalog, "Failed to copy memory from host d_stlp2z!\n", hipGetErrorString(err));
		fclose(cudalog);
		exit(EXIT_FAILURE);
	}
	//��3
	float* d_stlp3x = NULL;			err = hipMalloc((void **)&d_stlp3x, sizeSTL);
	if (err != hipSuccess) {
		fprintf(cudalog, "Failed to allocate device d_stlp3x!\n", hipGetErrorString(err));
		fclose(cudalog);
		exit(EXIT_FAILURE);
	}
	err = hipMemcpy(d_stlp3x, _stlp3x, sizeSTL, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		fprintf(cudalog, "Failed to copy memory from host d_stlp3x!\n", hipGetErrorString(err));
		fclose(cudalog);
		exit(EXIT_FAILURE);
	}

	float* d_stlp3y = NULL;			err = hipMalloc((void **)&d_stlp3y, sizeSTL);
	if (err != hipSuccess) {
		fprintf(cudalog, "Failed to allocate device d_stlp3y!\n", hipGetErrorString(err));
		fclose(cudalog);
		exit(EXIT_FAILURE);
	}
	err = hipMemcpy(d_stlp3y, _stlp3y, sizeSTL, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		fprintf(cudalog, "Failed to copy memory from host d_stlp3y!\n", hipGetErrorString(err));
		fclose(cudalog);
		exit(EXIT_FAILURE);
	}

	float* d_stlp3z = NULL;			err = hipMalloc((void **)&d_stlp3z, sizeSTL);
	if (err != hipSuccess) {
		fprintf(cudalog, "Failed to allocate device d_stlp3z!\n", hipGetErrorString(err));
		fclose(cudalog);
		exit(EXIT_FAILURE);
	}
	err = hipMemcpy(d_stlp3z, _stlp3z, sizeSTL, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		fprintf(cudalog, "Failed to copy memory from host d_stlp3z!\n", hipGetErrorString(err));
		fclose(cudalog);
		exit(EXIT_FAILURE);
	}

	//����GPU�ڴ� - �����
	bool* d_intersected = NULL;		err = hipMalloc((void **)&d_intersected, sizeSourceBool);
	if (err != hipSuccess) {
		fprintf(cudalog, "Failed to allocate device d_intersected!\n", hipGetErrorString(err));
		fclose(cudalog);
		exit(EXIT_FAILURE);
	}
	float* d_prot = NULL;			err = hipMalloc((void **)&d_prot, sizeSource);
	if (err != hipSuccess) {
		fprintf(cudalog, "Failed to allocate device d_prot!\n", hipGetErrorString(err));
		fclose(cudalog);
		exit(EXIT_FAILURE);
	}
	float* d_inter_x = NULL;		err = hipMalloc((void **)&d_inter_x, sizeSource);
	if (err != hipSuccess) {
		fprintf(cudalog, "Failed to allocate device d_inter_x!\n", hipGetErrorString(err));
		fclose(cudalog);
		exit(EXIT_FAILURE);
	}
	float* d_inter_y = NULL;		err = hipMalloc((void **)&d_inter_y, sizeSource);
	if (err != hipSuccess) {
		fprintf(cudalog, "Failed to allocate device d_inter_y!\n", hipGetErrorString(err));
		fclose(cudalog);
		exit(EXIT_FAILURE);
	}
	float* d_inter_z = NULL;		err = hipMalloc((void **)&d_inter_z, sizeSource);
	if (err != hipSuccess) {
		fprintf(cudalog, "Failed to allocate device d_inter_z!\n", hipGetErrorString(err));
		fclose(cudalog);
		exit(EXIT_FAILURE);
	}
	//�������������
	int* d_STLIndex = NULL;			err = hipMalloc((void **)&d_STLIndex, sizeSourceInt);
	if (err != hipSuccess)	{
		fprintf(cudalog, "Failed to allocate device d_STLIndex!\n", hipGetErrorString(err));
		fclose(cudalog);
		exit(EXIT_FAILURE);
	}
	/*
	//���������Ϣ����
	FILE* InputCheck;
	InputCheck = fopen("./SourceRay.txt", "w");
	for (int si = 0; si < _NumPoints; si++) {
		//if (_intersected[si]) fprintf(InputCheck, "ture ");
		//else fprintf(InputCheck, "false ");
		fprintf(InputCheck, "%f %f %f %f %f %f \n", _psourcex[si], _psourcey[si], _psourcez[si], _pdirx[si], _pdiry[si], _pdirz[si]);
	}
	fclose(InputCheck);
	//���������Ϣ����
	FILE* MeshCheck;
	MeshCheck = fopen("./Mesh.txt", "w");
	for (int si = 0; si < _NumSTL; si++) {
		//if (_intersected[si]) fprintf(InputCheck, "ture ");
		//else fprintf(InputCheck, "false ");
		fprintf(MeshCheck, "%f %f %f %f %f %f %f %f %f \n", _stlp1x[si], _stlp1y[si], _stlp1z[si], _stlp2x[si], _stlp2y[si], _stlp2z[si], _stlp3x[si], _stlp3y[si], _stlp3z[si]);
	}
	fclose(MeshCheck);
	*/




	//��GPU��ʼ��������
	int blocksPerGrid = (_NumPoints + threadsPerBlock - 1) / threadsPerBlock;
	fprintf(cudalog, "First, CUDA kernel will launch %d blocks of %d threads for Setting intersection results to Zero on GPU.\n", blocksPerGrid, threadsPerBlock);
	fclose(cudalog); cudalog = fopen("./cudalog_calculation.txt", "a");
	SetResultsZero <<< blocksPerGrid, threadsPerBlock >>>
		(_NumPoints,_NumSTL,d_intersected,d_prot,d_STLIndex,d_inter_x,d_inter_y,d_inter_z);

	//ֻ��GPU�����ҽ���
	//��������Block �� Thread��
	blocksPerGrid = (_NumSTL + threadsPerBlock - 1) / threadsPerBlock;
	fprintf(cudalog,"Second, CUDA kernel will launch %d blocks of %d threads for finding the intersection.\n", blocksPerGrid, threadsPerBlock);
	fclose(cudalog); cudalog = fopen("./cudalog_calculation.txt", "a");
	for (int si = 0; si < _NumPoints; si++) {
		float sx = _psourcex[si];
		float sy = _psourcey[si];
		float sz = _psourcez[si];
		float dx = _pdirx[si];
		float dy = _pdiry[si];
		float dz = _pdirz[si];
		FindInterSection_register <<< blocksPerGrid, threadsPerBlock >>> 
		(sx, sy, sz, dx, dy, dz,
			si,
			d_stlp1x, d_stlp1y, d_stlp1z,
			d_stlp2x, d_stlp2y, d_stlp2z,
			d_stlp3x, d_stlp3y, d_stlp3z,
			d_intersected, d_prot, d_STLIndex,
			d_inter_x, d_inter_y, d_inter_z,
			NumSTL);
		//��ɽ�����Ѱ
		if (si % 100 == 0) {
			fprintf(cudalog, "CUDA kernel finished %d Source Point of %d SourcePoints\n", si, _NumPoints);
			fclose(cudalog); cudalog = fopen("./cudalog_calculation.txt", "a");
		}
	}

	//��CUDA����������
	err = hipMemcpy(_intersected,d_intersected, sizeSourceBool, hipMemcpyDeviceToHost);
	err = hipMemcpy(_STLIndex, d_STLIndex, sizeSourceInt, hipMemcpyDeviceToHost);
	err = hipMemcpy(_prot, d_prot, sizeSource, hipMemcpyDeviceToHost);
	err = hipMemcpy(_inter_x, d_inter_x, sizeSource, hipMemcpyDeviceToHost);
	err = hipMemcpy(_inter_y, d_inter_y, sizeSource, hipMemcpyDeviceToHost);
	err = hipMemcpy(_inter_z, d_inter_z, sizeSource, hipMemcpyDeviceToHost);
	/*
	//���������Ϣ����
	FILE* ResultCheck;
	ResultCheck = fopen("./Intersection Results.txt", "w");
	for (int si = 0; si < _NumPoints; si++) {
		if (_intersected[si]) fprintf(ResultCheck, "ture ");
		else fprintf(ResultCheck, "false ");
		fprintf(ResultCheck, "%f %f %f %f \n", _prot[si], _inter_x[si], _inter_y[si], _inter_z[si]);
	}
	fclose(ResultCheck);
	*/
	fprintf(cudalog, "CUDA kernel finished computation��Next to free CUDA Memories\n", _NumPoints);
	fclose(cudalog); cudalog = fopen("./cudalog_calculation.txt", "a");
	err = hipFree(d_psourcex);	err = hipFree(d_psourcey);	err = hipFree(d_psourcez);
	err = hipFree(d_pdirx);	err = hipFree(d_pdiry);	err = hipFree(d_pdirz);
	err = hipFree(d_stlp1x);	err = hipFree(d_stlp1y);	err = hipFree(d_stlp1z);
	err = hipFree(d_stlp2x);	err = hipFree(d_stlp2y);	err = hipFree(d_stlp2z);
	err = hipFree(d_stlp3x);	err = hipFree(d_stlp3y);	err = hipFree(d_stlp3z);
	err = hipFree(d_intersected);
	err = hipFree(d_STLIndex);
	err = hipFree(d_prot);
	err = hipFree(d_inter_x);
	err = hipFree(d_inter_y);
	err = hipFree(d_inter_z);

	fprintf(cudalog, "CUDA Memory Cleaned\n", _NumPoints);
	fclose(cudalog); cudalog = fopen("./cudalog_calculation.txt", "a");
	fclose(cudalog);
}


